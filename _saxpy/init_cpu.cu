
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 2048 * 2048 // Number of elements in each vector

__global__ void saxpy(int * a, int * b, int * c)
{
  // Determine our unique global thread ID, so we know which element to process
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (int i = tid; i < N; i += stride)
    c[i] = 2 * a[i] + b[i];
}

// check error
inline hipError_t cudaCheck(hipError_t result){
	if (result!=hipSuccess){
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result==hipSuccess);
	}
	return result;
}

int main()
{
  int *a, *b, *c;

  int size = N * sizeof (int); // The total number of bytes per vector

  int deviceId;
  int numberOfSMs;
  
	cudaCheck(hipGetDevice(&deviceId));
  cudaCheck(hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId));

  // Allocate memory
  cudaCheck(hipMallocManaged(&a, size)); // both
  cudaCheck(hipMallocManaged(&b, size)); // both
  cudaCheck(hipMallocManaged(&c, size)); // both

  // Initialize memory
	for (int i = 0; i < N; ++i){
		a[i] = 2;
		b[i] = 1;
	}

	/*could be a windows problem but prefetching with cudaCheck defined on line 16
		gives error: "invalid device ordinal"
		try uncommenting lines 52-54 if you are on linux and run
	*/
//  cudaCheck(cudaMemPrefetchAsync(a, size, deviceId));
//  cudaCheck(cudaMemPrefetchAsync(b, size, deviceId));
//  cudaCheck(cudaMemPrefetchAsync(c, size, deviceId));
  
	int threads_per_block = 256;
  int number_of_blocks = numberOfSMs * 32;

  saxpy <<<number_of_blocks, threads_per_block>>>(a, b, c);

  cudaCheck(hipGetLastError());
  cudaCheck(hipDeviceSynchronize()); // Wait for the GPU to finish

  // Print out the first and last 5 values of c for a quality check
  for( int i = 0; i < 5; ++i )
    printf("c[%d] = %d, ", i, c[i]);
  printf ("\n");
  for( int i = N-5; i < N; ++i )
    printf("c[%d] = %d, ", i, c[i]);
  printf ("\n");

  // Free all our allocated memory
  cudaCheck(hipFree(a)); cudaCheck(hipFree(b)); cudaCheck(hipFree(c));
}
