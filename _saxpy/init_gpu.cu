
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 2048 * 2048 // Number of elements in each vector

__global__ void saxpy(int * a, int * b, int * c)
{
  // Determine our unique global thread ID, so we know which element to process
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (int i = tid; i < N; i += stride)
    c[i] = 2 * a[i] + b[i];
}

// init on gpu
__global__ void initWith(int value, int * a){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

	for (int i = tid; i< N; i += stride)
		a[i] = value;
}

// check error
inline hipError_t cudaCheck(hipError_t result){
	if (result!=hipSuccess){
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result==hipSuccess);
	}
	return result;
}

int main()
{
  int *a, *b, *c;

  int size = N * sizeof (int); // The total number of bytes per vector

  int deviceId;
  int numberOfSMs;

  cudaCheck(hipGetDevice(&deviceId));
  cudaCheck(hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId));

  // Allocate memory
  cudaCheck(hipMalloc(&a, size)); // gpu only
  cudaCheck(hipMalloc(&b, size)); // gpu only
  cudaCheck(hipMallocManaged(&c, size)); // both

  int threads_per_block = 256;
  int number_of_blocks = numberOfSMs * 32;

  // Initialize memory
	initWith <<<number_of_blocks, threads_per_block>>>(2, a);
	initWith <<<number_of_blocks, threads_per_block>>>(1, b);

  saxpy <<<number_of_blocks, threads_per_block>>>(a, b, c);

  cudaCheck(hipGetLastError());
  cudaCheck(hipDeviceSynchronize()); // Wait for the GPU to finish

  // Print out the first and last 5 values of c for a quality check
  for( int i = 0; i < 5; ++i )
    printf("c[%d] = %d, ", i, c[i]);
  printf ("\n");
  for( int i = N-5; i < N; ++i )
    printf("c[%d] = %d, ", i, c[i]);
  printf ("\n");

  // Free all our allocated memory
  cudaCheck(hipFree(a)); cudaCheck(hipFree(b)); cudaCheck(hipFree(c));
}
